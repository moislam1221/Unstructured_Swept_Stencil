using namespace std; 

#include "inttypes.h"
#include "global.h"
#include <vector>
#include <set>
#include <stdio.h>
#include <iostream>
#include <stdlib.h>
#include "helper.h"

int main (int argc, char * argv[]) 
{
    // Inputs
    uint32_t numIterations = 1;
    uint32_t N = 10;

    // Define solution
    uint32_t Ndofs = N*N;
    float *du0 = new float[Ndofs];
    float *du1 = new float[Ndofs];
    initializeToZeros(du0, Ndofs);
    initializeToZeros(du1, Ndofs);
    float *du0_d;
    float *du1_d;
    hipMalloc(&du0_d, sizeof(float) * Ndofs);
    hipMalloc(&du1_d, sizeof(float) * Ndofs);
    hipMemcpy(du0_d, du0, sizeof(float) * Ndofs, hipMemcpyHostToDevice);
    hipMemcpy(du1_d, du1, sizeof(float) * Ndofs, hipMemcpyHostToDevice);
 
    // Define rhs
    float *rhs = new float[Ndofs];
    initializeToOnes(rhs, Ndofs);
    float *rhs_d;
    hipMalloc(&rhs_d, sizeof(float) * Ndofs);
    hipMemcpy(rhs_d, rhs, sizeof(float) * Ndofs, hipMemcpyHostToDevice);

    // Perform iterations in global memory
    globalMemorySolve2D(du1_d, du0_d, N, numIterations); 

    /* */
    
    // Create the idxPtr and nodeNeighbors for 2D Structured
    uint32_t numEntries = (N-1)*(N-1)*5 + 4*(N-2)*4 + 4*3;
    uint32_t * idxPtr = new uint32_t[Ndofs+1];
    uint32_t * nodeNeighbors = new uint32_t[numEntries];
    construct2DConnectivity(idxPtr, nodeNeighbors, N);

    // Establish the partitioning for the first upper pyramidal stage
    vector<set<uint32_t>> arrayOfSeeds;
    set<uint32_t> seeds;
    // Fill up seeds
    // Seed 1
    seeds.insert(0);
    arrayOfSeeds.push_back(seeds);
    // Seed 2
    seeds.clear();
    seeds.insert(N);
    arrayOfSeeds.push_back(seeds);
    // Seed 3
    seeds.clear();
    seeds.insert(N*N-N);
    arrayOfSeeds.push_back(seeds);
    // Seed 4
    seeds.clear();
    seeds.insert(N*N-1);
    arrayOfSeeds.push_back(seeds);
 
    // Expand the seeds into the territories 
    uint32_t * subdomainOfNodes = new uint32_t[Ndofs];
    initializeToMaxInt(subdomainOfNodes, Ndofs);
    expand(arrayOfSeeds, subdomainOfNodes, Ndofs, idxPtr, nodeNeighbors);
    for (int i = 0; i < Ndofs; i++) {
        printf("subdomainOfNodes[%d] = %d\n", i, subdomainOfNodes[i]);
    }    

    // sharedMemorySolveUP<<<blocks, threadsPerBlock>>>
     

    // Perform iterations in shared memory using swept
    // sharedMemorySolve(); 

    // Print out timings

}
