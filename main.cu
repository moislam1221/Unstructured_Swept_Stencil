#include "inttypes.h"
#include "global.h"
#include "helper.h"

int main (int argc, char * argv[]) 
{
    // Inputs
    uint32_t numIterations = 1000;
    uint32_t N = 100;

    // Define solution
    uint32_t Ndofs = N*N;
    float *du0 = new float[Ndofs];
    float *du1 = new float[Ndofs];
    initializeToZeros(du0, Ndofs);
    initializeToZeros(du1, Ndofs);
    float *du0_d;
    float *du1_d;
    hipMalloc(&du0_d, sizeof(float) * Ndofs);
    hipMalloc(&du1_d, sizeof(float) * Ndofs);
    hipMemcpy(du0_d, du0, sizeof(float) * Ndofs, hipMemcpyHostToDevice);
    hipMemcpy(du1_d, du1, sizeof(float) * Ndofs, hipMemcpyHostToDevice);
 
    // Define rhs
    float *rhs = new float[Ndofs];
    initializeToOnes(rhs, Ndofs);
    float *rhs_d;
    hipMalloc(&rhs_d, sizeof(float) * Ndofs);
    hipMemcpy(rhs_d, rhs, sizeof(float) * Ndofs, hipMemcpyHostToDevice);

    // Perform iterations in global memory
    globalMemorySolve2D(du1_d, du0_d, N, numIterations); 

    // Perform iterations in shared memory using swept
    // sharedMemorySolve(); 

    // Print out timings

}
